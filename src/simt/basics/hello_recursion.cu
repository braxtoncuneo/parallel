
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

__device__ uint64_t fibonacci(uint64_t a, uint64_t b, uint64_t index) {
    if(index <= 0){
        return a;
    } else {
        return fibonacci(b,a+b,index-1);
    }
}

__global__ void hello(uint64_t index) {
    printf("The %ldth fibonacci number is %ld\n",index,fibonacci(0,1,index));
}

int main(int argc, char *argv[]) {
    uint64_t index = (argc>1) ? atoi(argv[1]) : 0;
    hello<<<1,1>>>(index);
    hipDeviceSynchronize();
}