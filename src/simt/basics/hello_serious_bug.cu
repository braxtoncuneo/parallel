
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <stdexcept>
#include <string>

void auto_throw(hipError_t status) {
    if(status != hipSuccess) {
        std::string message = "ERROR: '";
        message += hipGetErrorString(status);
        message +="'\n";
        throw std::runtime_error(message);
    }
}

__device__ uint64_t fibonacci(uint64_t index) {
    if(index <= 2){
        return 1;
    }
    uint64_t final_a = fibonacci(index-2);
    uint64_t final_b = fibonacci(index-1);
    return final_a + final_b;
}

__global__ void hello(uint64_t index) {
    printf("The %ldth fibonacci number is %ld\n",index,fibonacci(index));
}

int main(int argc, char *argv[]) {
    uint64_t index = (argc>1) ? atoi(argv[1]) : 0;
    hello<<<1,1>>>(index);
    auto_throw(hipDeviceSynchronize());
}