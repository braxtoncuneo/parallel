
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

void auto_throw(hipError_t status) {
    if(status != hipSuccess) {
        std::string message = "ERROR: '";
        message += hipGetErrorString(status);
        message +="'\n";
        throw std::runtime_error(message);
    }
}

__global__ void array_square(int* array, size_t size) {
    for(size_t i=threadIdx.x; i<size; i+=32){
        array[i] = array[i] * array[i];
    }
}

void print_array(int* array, size_t size) {
    for(size_t i=0; i<size; i++){
        if(i != 0){
            std::cout << ',';
        }
        std::cout << array[i];
    }
    std::cout << '\n';
}

int main(int argc, char *argv[]) {
    size_t size = (argc>1) ? atoi(argv[1]) : 0;

    int *cpu_array = new int[size];
    int *gpu_array;
    auto_throw(hipMalloc(&gpu_array,size*sizeof(int)));

    for(size_t i=0; i<size; i++){
        cpu_array[i] = i;
    }
    print_array(cpu_array,size);

    auto_throw(hipMemcpy(
        gpu_array,
        cpu_array,
        size*sizeof(int),
        hipMemcpyHostToDevice
    ));
    auto_throw(hipDeviceSynchronize());

    array_square<<<1,32>>>(gpu_array,size);
    auto_throw(hipDeviceSynchronize());

    auto_throw(hipMemcpy(
        cpu_array,
        gpu_array,
        size*sizeof(int),
        hipMemcpyDeviceToHost
    ));
    auto_throw(hipDeviceSynchronize());

    print_array(cpu_array,size);

    auto_throw(hipFree(gpu_array));
    delete[] cpu_array;
    return 0;
}